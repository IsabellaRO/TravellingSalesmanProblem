#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <iomanip> 
#include <vector>
#include <algorithm>
#include <random>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/random.h>

using namespace std;

__host__ __device__
double total_dist(int *order, double *mat, int N, int k) {
    double dist = 0.0;
    for(int i = 0; i < N; i++){
        if(i == N - 1){
            dist += mat[(order[(N*k)+N-1]*N)+order[(N*k)+0]];
        } else{
            dist += mat[(order[(N*k)+i]*N)+order[(N*k)+i+1]];
        }
    }
    return dist;
}

__host__ __device__
double troca_dist(double L, double *mat, int *order, int i, int j, int N, int k) {
    int c1 = order[(N*k)+i];
    int c2 = order[(N*k)+j];

    int jmais;
    (j == N-1 ? jmais = 0 : jmais = j+1);

    int jmenos;
    (j == 0 ? jmenos = N-1 : jmenos = j-1);
    
    int imais;
    (i == N-1 ? imais = 0 : imais = i+1);

    int imenos;
    (i == 0 ? imenos = N-1 : imenos = i-1);

	double new_L = L;
	if((i == 0 && j == N-1) || (i - j == 1)){
		new_L = new_L - mat[(order[(N*k)+jmenos]*N)+c2] + mat[(order[(N*k)+jmenos]*N)+c1];
		new_L = new_L - mat[(c1*N)+order[(N*k)+imais]] + mat[(c2*N)+order[(N*k)+imais]];
        return new_L;
	} else if((j == 0 && i == N-1) || (j - i == 1)){
		new_L = new_L - mat[(order[(N*k)+imenos]*N)+c1] + mat[(order[(N*k)+imenos]*N)+c2];
		new_L = new_L - mat[(c2*N)+order[(N*k)+jmais]] + mat[(c1*N)+order[(N*k)+jmais]];
        return new_L;
	} else {
        new_L = new_L - mat[(order[(N*k)+imenos]*N)+c1] + mat[(order[(N*k)+imenos]*N)+c2];
        new_L = new_L - mat[(order[(N*k)+jmenos]*N)+c2] + mat[(order[(N*k)+jmenos]*N)+c1];
        new_L = new_L - mat[(c1*N)+order[(N*k)+imais]] + mat[(c2*N)+order[(N*k)+imais]];
        new_L = new_L - mat[(c2*N)+order[(N*k)+jmais]] + mat[(c1*N)+order[(N*k)+jmais]];
        return new_L;
	}
}

struct raw_access {
    int *order;
    int *new_order;
    int *solutions;
    int seed;
    int debug;
    int N;
    double *L;
    bool *in_tour;
    double *mat;

    raw_access (int *order, int *new_order, int *solutions, int seed, int debug, int N, double *L, bool *in_tour, double *mat) : order(order), new_order(new_order), solutions(solutions), seed(seed), debug(debug), N(N), L(L), in_tour(in_tour), mat(mat) {};
    
    __host__ __device__
    double operator()(const int &i) {
        int counter = 0;
        
        thrust::default_random_engine generator(seed);

        while(counter < N){ // Criar tour aleatório
            for(int k = 0; k < N; k++){
                
                generator.discard(i);
                thrust::uniform_real_distribution<float> distribution(0, 1);
                double decision = distribution(generator);
                
                if(decision >= 0.5 && !in_tour[(i*N)+k]){ // entra no tour
                    order[(i*N)+counter] = k;
                    in_tour[(i*N)+k] = true;
                    counter++;
                }
            }
        }

        for(int h = 0; h < N; h++){
            for(int j = 0; j < N; j++){
                if(h != j){
                    for(int l = 0; l < N; l++){
                        new_order[(i*N)+l] = order[(i*N)+l];
                    }
                    
                    new_order[(i*N)+j] = order[(i*N)+h];
                    new_order[(i*N)+h] = order[(i*N)+j];
                    
                    L[i] = total_dist(order, mat, N, i);
                    double new_L = troca_dist(L[i], mat, order, h, j, N, i);
                    
                    if(new_L < L[i]){
                        order[(i*N)+h] = new_order[(i*N)+h];
                        order[(i*N)+j] = new_order[(i*N)+j];
                        L[i] = new_L;
                        h = 0;
                        j = 0;
                    }
                }
            }
        }

        // ADICIONA SOLUÇÃO ATUAL NAS SOLUÇÕES
        for(int k = 0; k < N; k++){
            solutions[(i*N)+k] = order[(i*N)+k];
        }

        return L[i];
    }
};

int main() {
    // PEGA SEED E DEBUG OU SETA COMO DEFAULT
    int seed = 10;
    char *SEED(getenv("SEED"));
    if (SEED != NULL){
        seed = atoi(SEED);
    }
    int debug = 0;
    char *DEBUG(getenv("DEBUG"));
    if (DEBUG != NULL){
        debug = atoi(DEBUG);
    }
    
    // LÊ ENTRADA
    int N;
    cin >> N;
    
    thrust::device_vector<int> solutions(10*N*N); // tours possíveis
    thrust::device_vector<double> distancias(10*N); // distancias de cada tour
    thrust::device_vector<double> L(10*N, 0.0); // um pra cada thread
    int O = 0;

    thrust::device_vector<double> X(N);
    thrust::device_vector<double> Y(N);
    
    for(int i = 0; i < N; i++){
        double x, y;
        cin >> x >> y;
        X[i] = x;
        Y[i] = y;
    }

    thrust::device_vector<double> mat(N*N);
    for(int i = 0; i < N; i++){
        double dist;
        for(int j = i; j < N; j++){
            double dx = X[i] - X[j];
            double dy = Y[i] - Y[j];
            dist = sqrt(pow(dx, 2) + pow(dy, 2));
            mat[i*N + j] = dist;
            mat[j*N + i] = dist;
        }
    }

    thrust::device_vector<bool> in_tour (10*N*N, false);
    thrust::device_vector<int> order(10*N*N);
    thrust::device_vector<int> new_order(10*N*N);

    thrust::counting_iterator<int> iter(0);
    raw_access ra(thrust::raw_pointer_cast(order.data()), thrust::raw_pointer_cast(new_order.data()), thrust::raw_pointer_cast(solutions.data()), seed, debug, N, thrust::raw_pointer_cast(L.data()), thrust::raw_pointer_cast(in_tour.data()), thrust::raw_pointer_cast(mat.data()) );
    thrust::transform(iter, iter+distancias.size(), distancias.begin(), ra);

    double menordist = 100000;
    int position = -1;
    for(int i = 0; i < 10*N; i++){
        if(distancias[i] < menordist){
            menordist = distancias[i];
            position = i;
        }
    }


    //thrust::copy(solutions.begin(), solutions.end(), solucoes);
    thrust::host_vector<int> solucoes(10*N);
    solucoes = solutions;

    thrust::host_vector<int> ordens(10*N*N);
    ordens = order;

    thrust::host_vector<int> solucao_final(N);
    for(int i = 0; i < N; i++){
        solucao_final[i] = solucoes[(position*N)+i];
    }

    if(debug == 1){
        for(int k = 0; k < 10*N; k++){
            cerr << "local " << distancias[k];
            for (int m = 0; m < N; m++){
                cerr << " " << ordens[(k*N)+m];
            }
            cerr << "\n";
        }
    }

    cout << menordist << " " << O << endl;
    for(int i = 0; i < N; i++){
        cout << solucao_final[i] << " ";
    }
    cout << endl;

    return 0;
}

